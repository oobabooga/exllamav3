#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_fp16.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;
#include "../../util.h"
#include "../../util.cuh"
#include "../../ptx.cuh"
#include "../exl3_gemm_kernel.cuh"
#include "exl3_comp_unit_8.cuh"

fp_exl3_gemm_kernel tfp_exl3_gemm_kernel_fp32_b8[] = {
    nullptr,
    exl3_gemm_kernel<8, true, EXL3_GEMM_SHAPE_1>,
    exl3_gemm_kernel<8, true, EXL3_GEMM_SHAPE_2>,
    exl3_gemm_kernel<8, true, EXL3_GEMM_SHAPE_3>,
    exl3_gemm_kernel<8, true, EXL3_GEMM_SHAPE_4>
};

fp_exl3_gemm_kernel tfp_exl3_gemm_kernel_fp16_b8[] = {
    nullptr,
    exl3_gemm_kernel<8, false, EXL3_GEMM_SHAPE_1>,
    exl3_gemm_kernel<8, false, EXL3_GEMM_SHAPE_2>,
    exl3_gemm_kernel<8, false, EXL3_GEMM_SHAPE_3>,
    exl3_gemm_kernel<8, false, EXL3_GEMM_SHAPE_4>
};
