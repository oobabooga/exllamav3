#include "hip/hip_runtime.h"
#include "quantize.cuh"
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_fp16.h>
#include "../util.h"
#include "../util.cuh"

__device__ inline half hreduce(half2 x)
{
    return __hadd(__low2half(x), __high2half(x));
}

__device__ inline float shuffle_had_fx32(float v, int lane_id)
{
    for (int i = 1; i < 32; i <<= 1)
    {
        float pv = __shfl_xor_sync(0xffffffff, v, i);
        uint32_t* vi = reinterpret_cast<uint32_t*>(&v);
        int32_t sfm = -static_cast<int16_t>(lane_id & i) >> 31;
        *vi ^= (sfm & 0x80000000);
        v = v + pv;
    }
    return v;
}

__device__ inline half2 shuffle_had_h2x32(half2 v, int lane_id)
{
    for (int i = 1; i < 32; i <<= 1)
    {
        half2 pv = __shfl_xor_sync(0xffffffff, v, i);
        uint32_t* vi = reinterpret_cast<uint32_t*>(&v);
        int32_t sfm = -static_cast<int16_t>(lane_id & i) >> 31;
        *vi ^= (sfm & 0x80008000);
        v = __hadd2(v, pv);
    }
    return v;
}

__global__ __launch_bounds__(32)
void hadh_r_128_kernel
(
    const half* __restrict__ input_ptr,
    half* __restrict__ output_ptr,
    const half* __restrict__ pre_scale,
    const half* __restrict__ post_scale
)
{
    int t = threadIdx.x;
    input_ptr += gridDim.y * 128 * blockIdx.x + blockIdx.y * 128;
    output_ptr += gridDim.y * 128 * blockIdx.x + blockIdx.y * 128;

    // Load
    half4 v = ((half4*) input_ptr)[t];

    // Prescale
    if (pre_scale)
    {
        pre_scale += blockIdx.y * 128;
        half4 s = ((half4*) pre_scale)[t];
        v.x = __h2div(v.x, s.x);
        v.y = __h2div(v.y, s.y);
    }

    // 4 element had
    half2 vxpp = v.x;
    half2 vxpn = h2xor(vxpp, 0x80000000);
    half2 vypp = v.y;
    half2 vypn = h2xor(vypp, 0x80000000);
    half h0 = hreduce(__hadd2(vxpp, vypp));
    half h1 = hreduce(__hadd2(vxpn, vypn));
    half h2 = hreduce(__hsub2(vxpp, vypp));
    half h3 = hreduce(__hsub2(vxpn, vypn));
    v.x = __halves2half2(h0, h1);
    v.y = __halves2half2(h2, h3);

    // 32 element had, warp shuffle
    v.x = shuffle_had_h2x32(v.x, t);
    v.y = shuffle_had_h2x32(v.y, t);

    // Rescale by 1/sqrt(128)
    half2 f = __halves2half2(__float2half_rn(0.088388347648), __float2half_rn(0.088388347648));
    v.x = __hmul2(v.x, f);
    v.y = __hmul2(v.y, f);

    // Postscale
    if (post_scale)
    {
        post_scale += blockIdx.y * 128;
        half4 s = ((half4*) post_scale)[t];
        v.x = __h2div(v.x, s.x);
        v.y = __h2div(v.y, s.y);
    }

    // Store
    ((half4*) output_ptr)[t] = v;
}

__global__ __launch_bounds__(32)
void had_hf_r_128_kernel
(
    const half* __restrict__ input_ptr,
    half* __restrict__ output_ptr,
    const half* __restrict__ pre_scale,
    const half* __restrict__ post_scale,
    float r_scale
)
{
    int t = threadIdx.x;
    input_ptr += gridDim.y * 128 * blockIdx.x + blockIdx.y * 128;
    output_ptr += gridDim.y * 128 * blockIdx.x + blockIdx.y * 128;

    // Load
    half4 v = ((half4*) input_ptr)[t];

    // Pre scale
    if (pre_scale)
    {
        int i = blockIdx.y * 32 + t;
        half4 scales = ((half4*) pre_scale)[i];
        v.x = __hmul2(v.x, scales.x);
        v.y = __hmul2(v.y, scales.y);
    }

    // 4 element had
    float v0 = __half2float(__low2half(v.x));
    float v1 = __half2float(__high2half(v.x));
    float v2 = __half2float(__low2half(v.y));
    float v3 = __half2float(__high2half(v.y));
    float h0 = v0 + v1 + v2 + v3;
    float h1 = v0 - v1 + v2 - v3;
    float h2 = v0 + v1 - v2 - v3;
    float h3 = v0 - v1 - v2 + v3;

    // 32 element had, warp shuffle
    h0 = shuffle_had_fx32(h0, t) * r_scale;
    h1 = shuffle_had_fx32(h1, t) * r_scale;
    h2 = shuffle_had_fx32(h2, t) * r_scale;
    h3 = shuffle_had_fx32(h3, t) * r_scale;
    v.x = __floats2half2_rn(h0, h1);
    v.y = __floats2half2_rn(h2, h3);

    // Post scale
    if (post_scale)
    {
        int i = blockIdx.y * 32 + t;
        half4 scales = ((half4*) post_scale)[i];
        v.x = __hmul2(v.x, scales.x);
        v.y = __hmul2(v.y, scales.y);
    }

    // Store
    ((half4*) output_ptr)[t] = v;
}

__global__ __launch_bounds__(32)
void had_ff_r_128_kernel
(
    const float* __restrict__ input_ptr,
    float* __restrict__ output_ptr,
    const half* __restrict__ pre_scale,
    const half* __restrict__ post_scale,
    float r_scale
)
{
    int t = threadIdx.x;
    input_ptr += gridDim.y * 128 * blockIdx.x + blockIdx.y * 128;
    output_ptr += gridDim.y * 128 * blockIdx.x + blockIdx.y * 128;

    // Load
    float4 v = ((float4*) input_ptr)[t];

    // Pre scale
    if (pre_scale)
    {
        int i = blockIdx.y * 32 + t;
        half4 scales = ((half4*) pre_scale)[i];
        v.x *= __low2float(scales.x);
        v.y *= __high2float(scales.x);
        v.z *= __low2float(scales.y);
        v.w *= __high2float(scales.y);
    }

    // 4 element had
    float v0 = v.x;
    float v1 = v.y;
    float v2 = v.z;
    float v3 = v.w;
    float h0 = v0 + v1 + v2 + v3;
    float h1 = v0 - v1 + v2 - v3;
    float h2 = v0 + v1 - v2 - v3;
    float h3 = v0 - v1 - v2 + v3;

    // 32 element had, warp shuffle
    v.x = shuffle_had_fx32(h0, t) * r_scale;
    v.y = shuffle_had_fx32(h1, t) * r_scale;
    v.z = shuffle_had_fx32(h2, t) * r_scale;
    v.w = shuffle_had_fx32(h3, t) * r_scale;

    // Post scale
    if (post_scale)
    {
        int i = blockIdx.y * 32 + t;
        half4 scales = ((half4*) post_scale)[i];
        v.x *= __low2float(scales.x);
        v.y *= __high2float(scales.x);
        v.z *= __low2float(scales.y);
        v.w *= __high2float(scales.y);
    }

    // Store
    ((float4*) output_ptr)[t] = v;
}

/*
Compute y = (x.view(-1, 128) @ had_128).view(x.shape)
Works inplace if y == x
x and y must be same dtype, either float16 or float32
*/
void had_r_128
(
    const at::Tensor& input,
    const at::Tensor& output,
    const c10::optional<at::Tensor>& pre_scale,
    const c10::optional<at::Tensor>& post_scale,
    float scale
)
{
    const at::cuda::OptionalCUDAGuard device_guard(input.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

    TORCH_CHECK_SHAPES_FULL(input, output);
    TORCH_CHECK_DIM(input, 2);
    TORCH_CHECK_DIV(input, 1, 128);
    int rows = input.size(0);
    int cols = input.size(1);

    int blocks = cols / 128;
    float r_scale = scale * 0.088388347648f; // scale / sqrt(128)

    dim3 blockDim(32);
    dim3 gridDim(rows, blocks);

    if (input.dtype() == at::kHalf)
    {
        TORCH_CHECK_DTYPE(output, kHalf);
        had_hf_r_128_kernel<<<gridDim, blockDim, 0, stream>>>
        (
            (const half*) input.data_ptr(),
            (half*) output.data_ptr(),
            (const half*) OPTPTR(pre_scale),
            (const half*) OPTPTR(post_scale),
            r_scale
        );
    }

    else if (input.dtype() == at::kFloat)
    {
        TORCH_CHECK_DTYPE(output, kFloat);
        had_ff_r_128_kernel<<<gridDim, blockDim, 0, stream>>>
        (
            (const float*) input.data_ptr(),
            (float*) output.data_ptr(),
            (const half*) OPTPTR(pre_scale),
            (const half*) OPTPTR(post_scale),
            r_scale
        );
    }

    else TORCH_CHECK(false, "unsupported datatype");
}